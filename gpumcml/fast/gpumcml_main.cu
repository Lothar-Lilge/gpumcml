/////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//   GPU-based Monte Carlo simulation of photon migration in multi-layered media (GPUMCML)
//   Copyright (C) 2009
//	
//	 Some documentation is available for GPUMCML and should have been distributed along 
//	 with this source code. If that is not the case: Documentation, source code and executables
//	 for GPUMCML are available for download on our webpage:
//   http://code.google.com/p/gpumcml 
// 
//	 http://www.atomic.physics.lu.se/Biophotonics
//	 or, directly
//	 http://www.atomic.physics.lu.se/fileadmin/atomfysik/Biophotonics/Software/CUDAMCML.zip
//
//	 We encourage the use, and modification of this code, and hope it will help 
//	 users/programmers to utilize the power of GPGPU for their simulation needs. While we
//	 don't have a scientifc publication describing this code, we would very much appreciate
//	 if you cite our original GPGPU Monte Carlo letter (on which GPUMCML is based) if you 
//	 use this code or derivations thereof for your own scientifc work:
//	 E. Alerstam, T. Svensson and S. Andersson-Engels, "Parallel computing with graphics processing
//	 units for high-speed Monte Carlo simulations of photon migration", Journal of Biomedical Optics
//	 Letters, 13(6) 060504 (2008).
//
//	 To compile and run this code, please visit www.nvidia.com and download the necessary 
//	 CUDA Toolkit and SKD. We also highly recommend the Visual Studio wizard 
//	 (available at:http://forums.nvidia.com/index.php?showtopic=69183) 
//	 if you use Visual Studio 2005 
//	 (The express edition is available for free at: http://www.microsoft.com/express/2005/). 
//  	
//   This code is distributed under the terms of the GNU General Public Licence (see below). 
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	 
*   This file is part of GPUMCML.
* 
*   GPUMCML is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   GPUMCML is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with GPUMCML.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <float.h> //for FLT_MAX 
#include <stdio.h>

#include <hip/hip_runtime.h>

#ifdef _WIN32 
#include "gpumcml_io.c"
#include "cutil-win32/cutil.h"
#else 
#include <cutil.h>
#endif

#ifdef _WIN32 
#include "cutil-win32/multithreading.h"
#else
#include "multithreading.h"
#endif

#include "gpumcml.h"
#include "gpumcml_kernel.h"

#include "gpumcml_kernel.cu"
#include "gpumcml_mem.cu"

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
//   Supports multiple GPUs by allowing multiple host threads to launch kernel
//   Each thread calls RunGPUi with its own HostThreadState parameters
//////////////////////////////////////////////////////////////////////////////
static CUT_THREADPROC RunGPUi(HostThreadState *hstate)
{
  SimState *HostMem = &(hstate->host_sim_state);
  SimState DeviceMem;
  GPUThreadStates tstates;
  // total number of threads in the grid
  UINT32 n_threads = hstate->n_tblks * NUM_THREADS_PER_BLOCK;
  hipError_t cudastat;

  CUDA_SAFE_CALL( hipSetDevice(hstate->dev_id) );

  // Init the remaining states.
  InitSimStates(HostMem, &DeviceMem, &tstates, hstate->sim, n_threads);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitSimStates (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  InitDCMem(hstate->sim, hstate->A_rz_overflow);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitDCMem (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(hstate->n_tblks);

  int k_smem_sz = 0;
#ifdef USE_32B_ELEM_FOR_ARZ_SMEM
  // This piece of shared memory is for overflow handling.
  k_smem_sz = NUM_THREADS_PER_BLOCK * sizeof(UINT32);
#endif

  // Initialize the remaining thread states.
  InitThreadState<<<dimGrid,dimBlock>>>(tstates);
  CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
  cudastat=hipGetLastError(); // Check if there was an error
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in InitThreadState (%i): %s\n",
      hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

#ifdef USE_TRUE_CACHE
  // Configure the L1 cache for Fermi.
  if (hstate->sim->ignoreAdetection == 1)
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<1>), hipFuncCachePreferL1);
  }
  else
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<0>), hipFuncCachePreferL1);
  }
#endif

  for (int i = 1; *HostMem->n_photons_left > 0; ++i)
  {
    // Run the kernel.
    if (hstate->sim->ignoreAdetection == 1)
    {
      MCMLKernel<1><<<dimGrid, dimBlock, k_smem_sz>>>(DeviceMem, tstates);
    }
    else
    {
      MCMLKernel<0><<<dimGrid, dimBlock, k_smem_sz>>>(DeviceMem, tstates);
    }
    // Wait for all threads to finish.
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    // Check if there was an error
    cudastat = hipGetLastError();
    if (cudastat)
    {
      fprintf(stderr, "[GPU %u] failure in MCMLKernel (%i): %s.\n",
        hstate->dev_id, cudastat, hipGetErrorString(cudastat));
      FreeHostSimState(HostMem);
      FreeDeviceSimStates(&DeviceMem, &tstates);
      exit(1); 
    }

    // Copy the number of photons left from device to host.
    CUDA_SAFE_CALL( hipMemcpy(HostMem->n_photons_left,
      DeviceMem.n_photons_left, sizeof(unsigned int),
      hipMemcpyDeviceToHost) );

    printf("[GPU %u] batch %5d, number of photons left %10u\n",
      hstate->dev_id, i, *(HostMem->n_photons_left));
  }

  // Sum the multiple copies of A_rz in the global memory.
  sum_A_rz<<<30, 128>>>(DeviceMem.A_rz);
  // Wait for all threads to finish.
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  // Check if there was an error
  cudastat = hipGetLastError();
  if (cudastat)
  {
    fprintf(stderr, "[GPU %u] failure in sum_A_rz (%i): %s.\n",
        hstate->dev_id, cudastat, hipGetErrorString(cudastat));
    FreeHostSimState(HostMem);
    FreeDeviceSimStates(&DeviceMem, &tstates);
    exit(1); 
  }

  printf("[GPU %u] simulation done!\n", hstate->dev_id);

  CopyDeviceToHostMem(HostMem, &DeviceMem, hstate->sim, n_threads);
  FreeDeviceSimStates(&DeviceMem, &tstates);
  // We still need the host-side structure.
}

//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
static void DoOneSimulation(int sim_id, SimulationStruct* simulation,
                            HostThreadState* hstates[], UINT32 num_GPUs,
                            UINT64 *x, UINT32 *a)
{
  printf("\n------------------------------------------------------------\n");
  printf("        Simulation #%d\n", sim_id);
  printf("        - number_of_photons = %u\n", simulation->number_of_photons);

  // Compute GPU-specific constant parameters.
  UINT32 A_rz_overflow = 0;
  // We only need it if we care about A_rz.
#if !defined(USE_TRUE_CACHE) && defined(USE_32B_ELEM_FOR_ARZ_SMEM)
  if (! simulation->ignoreAdetection)
  {
    A_rz_overflow = compute_Arz_overflow_count(simulation->start_weight,
        simulation->layers, simulation->n_layers, NUM_THREADS_PER_BLOCK);
    printf("        - A_rz_overflow = %u\n", A_rz_overflow);
  }
#endif

  printf("------------------------------------------------------------\n\n");

  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Start the timer.
  hipEventRecord(start,0);

  // Distribute all photons among GPUs.
  unsigned int n_photons_per_GPU = simulation->number_of_photons / num_GPUs;

  // For each GPU, init the host-side structure.
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    hstates[i]->sim = simulation;
    hstates[i]->A_rz_overflow = A_rz_overflow;

    SimState *hss = &(hstates[i]->host_sim_state);

    // number of photons responsible 
    hss->n_photons_left = (UINT32*)malloc(sizeof(UINT32));
    // The last GPU may be responsible for more photons if the
    // distribution is uneven.
    *(hss->n_photons_left) = (i == num_GPUs-1) ?
      simulation->number_of_photons - (num_GPUs-1) * n_photons_per_GPU :
    n_photons_per_GPU;
  }

  // Launch a dedicated host thread for each GPU.
  CUTThread hthreads[MAX_GPU_COUNT];
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    hthreads[i] = cutStartThread((CUT_THREADROUTINE)RunGPUi, hstates[i]);
  }

  // Wait for all host threads to finish.
  cutWaitForThreads(hthreads, num_GPUs);


  // Check any of the threads failed.
  int failed = 0;
  for (UINT32 i = 0; i < num_GPUs && !failed; ++i)
  {
    if (hstates[i]->host_sim_state.n_photons_left == NULL) failed = 1;
  }

  if (!failed)
  {
    // Sum the results to hstates[0].
    SimState *hss0 = &(hstates[0]->host_sim_state);
    for (UINT32 i = 1; i < num_GPUs; ++i)
    {
      SimState *hssi = &(hstates[i]->host_sim_state);

      // A_rz
      int size = simulation->det.nr * simulation->det.nz;
      for (int j = 0; j < size; ++j)
      {
        hss0->A_rz[j] += hssi->A_rz[j];
      }

      // Rd_ra
      size = simulation->det.na * simulation->det.nr;
      for (int j = 0; j < size; ++j)
      {
        hss0->Rd_ra[j] += hssi->Rd_ra[j];
      }

      // Tt_ra
      size = simulation->det.na * simulation->det.nr;
      for (int j = 0; j < size; ++j)
      {
        hss0->Tt_ra[j] += hssi->Tt_ra[j];
      }
    }

    // End the timer.
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    // Compute the execution time.
    hipEventElapsedTime(&elapsedTime, start, stop);
    // Convert to seconds.
    elapsedTime /= 1000.0;
    printf("\n*** Simulation time: %.3f sec\n\n", elapsedTime);

    Write_Simulation_Results(hss0, simulation, elapsedTime);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Free SimState structs.
  for (UINT32 i = 0; i < num_GPUs; ++i)
  {
    FreeHostSimState(&(hstates[i]->host_sim_state));
  }
}

//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
int main(int argc, char* argv[])
{
  char* filename = NULL;
  UINT64 seed = (UINT64) time(NULL);
  int ignoreAdetection = 0;
  UINT32 num_GPUs = 1;

  SimulationStruct* simulations;
  int n_simulations;

  int i;

  // Parse command-line arguments.
  if (interpret_arg(argc, argv, &filename,
    &seed, &ignoreAdetection, &num_GPUs))
  {
    usage(argv[0]);
    return 1;
  }

  // Determine the number of GPUs available.
  int dev_count;
  CUDA_SAFE_CALL( hipGetDeviceCount(&dev_count) );
  if (dev_count <= 0)
  {
    fprintf(stderr, "No GPU available. Quit.\n");
    return 1;
  }

  // Make sure we do not use more than what we have.
  if (num_GPUs > dev_count)
  {
    printf("The number of GPUs specified (%u) is more than "
      "what is available (%d)!\n", num_GPUs, dev_count);
    num_GPUs = (UINT32)dev_count;
  }

  // Output the execution configuration.
  printf("\n====================================\n");
  printf("EXECUTION MODE:\n");
  printf("  ignore A-detection:      %s\n",
    ignoreAdetection ? "YES" : "NO");
  printf("  seed:                    %llu\n", seed);
  printf("  # of GPUs:               %u\n", num_GPUs);
  printf("====================================\n\n");

  // Read the simulation inputs.
  n_simulations = read_simulation_data(filename, &simulations,
    ignoreAdetection);
  if(n_simulations == 0)
  {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  }
  printf("Read %d simulations\n\n",n_simulations);

  // Allocate one host thread state for each GPU.
  HostThreadState* hstates[MAX_GPU_COUNT];
  hipDeviceProp_t props;
  int n_threads = 0;    // total number of threads for all GPUs
  for (i = 0; i < num_GPUs; ++i)
  {
    hstates[i] = (HostThreadState*)malloc(sizeof(HostThreadState));

    // Set the GPU ID.
    hstates[i]->dev_id = i;

    // Get the GPU properties.
    CUDA_SAFE_CALL( hipGetDeviceProperties(&props, hstates[i]->dev_id) );
    printf("[GPU %u] \"%s\" with Compute Capability %d.%d (%d SMs)\n",
        i, props.name, props.major, props.minor, props.multiProcessorCount);

    // Validate the GPU compute capability.
    int cc = props.major * 10 + props.minor;
    if (cc < CUDA_ARCH)
    {
      fprintf(stderr, "\nGPU %u does not meet the Compute Capability "
          "this program requires (%d)! Abort.\n\n", i, CUDA_ARCH);
      exit(1);
    }

    // We launch one thread block for each SM on this GPU.
    hstates[i]->n_tblks = props.multiProcessorCount;

    n_threads += hstates[i]->n_tblks * NUM_THREADS_PER_BLOCK;
  }

  // Allocate and initialize RNG seeds (for all threads on all GPUs).
  UINT64 *x = (UINT64*)malloc(n_threads * sizeof(UINT64));
  UINT32 *a = (UINT32*)malloc(n_threads * sizeof(UINT32));
  if (init_RNG(x, a, n_threads, "safeprimes_base32.txt", seed)) return 1;
  printf("\nUsing the MWC random number generator ...\n");

  // Assign these seeds to each host thread state.
  int ofst = 0;
  for (i = 0; i < num_GPUs; ++i)
  {
    SimState *hss = &(hstates[i]->host_sim_state);
    hss->x = &x[ofst];
    hss->a = &a[ofst];

    ofst += hstates[i]->n_tblks * NUM_THREADS_PER_BLOCK;
  }

  //perform all the simulations
  for(i=0;i<n_simulations;i++)
  {
    // Run a simulation
    DoOneSimulation(i, &simulations[i], hstates, num_GPUs, x, a);
  }

  // Free host thread states.
  for (i = 0; i < num_GPUs; ++i) free(hstates[i]);

  // Free the random number seed arrays.
  free(x); free(a);

  FreeSimulationStruct(simulations, n_simulations);

  return 0; 
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

