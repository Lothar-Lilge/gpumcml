#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//   GPU-based Monte Carlo simulation of photon migration in multi-layered media (GPU-MCML)
//   Copyright (C) 2009
//	
//   || DEVELOPMENT TEAM: 
//   --------------------------------------------------------------------------------------------------
//   Erik Alerstam, David Han, and William C. Y. Lo
//   
//   This code is the result of the collaborative efforts between 
//   Lund University and the University of Toronto.  
//
//   || DOCUMENTATION AND USER MANUAL: 
//   --------------------------------------------------------------------------------------------------
//	 Detailed "Wiki" style documentation is being developed for GPU-MCML 
//   and will be available on our webpage soon:
//   http://code.google.com/p/gpumcml 
// 
//   || NEW FEATURES: 
//   --------------------------------------------------------------------------------------------------
//    - Supports the Fermi GPU architecture 
//    - Multi-GPU execution 
//    - Automatic selection of optimization parameters  
//    - Backward compatible on pre-Fermi graphics cards
//    - Supports linux and Windows environment (Visual Studio)
//   
//   || PREVIOUS WORK: 
//   --------------------------------------------------------------------------------------------------
//	 This code is the fusion of our earlier, preliminary implementations and combines the best features 
//   from each implementation.  
//
//   W. C. Y. Lo, T. D. Han, J. Rose, and L. Lilge, "GPU-accelerated Monte Carlo simulation for photodynamic
//   therapy treatment planning," in Proc. of SPIE-OSA Biomedical Optics, vol. 7373.
//   
//   and 
//
//   http://www.atomic.physics.lu.se/biophotonics/our_research/monte_carlo_simulations/gpu_monte_carlo/
//	 E. Alerstam, T. Svensson and S. Andersson-Engels, "Parallel computing with graphics processing
//	 units for high-speed Monte Carlo simulations of photon migration", Journal of Biomedical Optics
//	 Letters, 13(6) 060504 (2008).
//
//   || CITATION: 
//   --------------------------------------------------------------------------------------------------
//	 We encourage the use, and modification of this code, and hope it will help 
//	 users/programmers to utilize the power of GPGPU for their simulation needs. While we
//	 don't have a scientific publication describing this code yet, we would very much appreciate it
//	 if you cite our original papers above if you use this code or derivations 
//   thereof for your own scientific work
//
//	 To compile and run this code, please visit www.nvidia.com and download the necessary 
//	 CUDA Toolkit, SDK, and Developer Drivers 
//
//	 If you use Visual Studio, the express edition is available for free at 
//   http://www.microsoft.com/express/Downloads/). 
//  	
//   This code is distributed under the terms of the GNU General Public Licence (see below). 
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	 
*   This file is part of GPUMCML.
* 
*   GPUMCML is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   GPUMCML is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with GPUMCML.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <float.h> //for FLT_MAX 
#include <stdio.h>

#include <hip/hip_runtime.h>

#ifdef _WIN32 
#include "gpumcml_io.c"
#include "cutil-win32/cutil.h"
#else 
#include <cutil.h>
#endif

#include "gpumcml.h"
#include "gpumcml_kernel.h"

#include "gpumcml_kernel.cu"
#include "gpumcml_mem.cu"

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////
//   Supports 1 GPU only
//   Calls RunGPU with HostThreadState parameters
//////////////////////////////////////////////////////////////////////////////
static void RunGPUi(HostThreadState *hstate)
{
  SimState *HostMem = &(hstate->host_sim_state);
  SimState DeviceMem;
  GPUThreadStates tstates;

  hipError_t cudastat;

  // Init the remaining states.
  InitSimStates(HostMem, &DeviceMem, &tstates, hstate->sim);

  InitDCMem(hstate->sim);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);

  // Initialize the remaining thread states.
  InitThreadState<<<dimGrid,dimBlock>>>(tstates);

  // Configure the L1 cache for Fermi.
#ifdef USE_TRUE_CACHE
  if (hstate->sim->ignoreAdetection == 1)
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<1>), hipFuncCachePreferL1);
  }
  else
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(MCMLKernel<0>), hipFuncCachePreferL1);
  }
#endif

  for (int i = 1; *HostMem->n_photons_left > 0; ++i)
  {
    // Run the kernel.
    if (hstate->sim->ignoreAdetection == 1)
    {
      MCMLKernel<1><<<dimGrid, dimBlock>>>(DeviceMem, tstates);
    }
    else
    {
      MCMLKernel<0><<<dimGrid, dimBlock>>>(DeviceMem, tstates);
    }

    // Check if there was an error
    cudastat = hipGetLastError();
    if (cudastat)
    {
      fprintf(stderr, "[GPU] failure in MCMLKernel (%i): %s.\n",cudastat, hipGetErrorString(cudastat));
      FreeHostSimState(HostMem);
      FreeDeviceSimStates(&DeviceMem, &tstates);
      exit(1); 
    }

    // Copy the number of photons left from device to host.
    CUDA_SAFE_CALL( hipMemcpy(HostMem->n_photons_left,
      DeviceMem.n_photons_left, sizeof(unsigned int),
      hipMemcpyDeviceToHost) );

    printf("[GPU] batch %5d, number of photons left %10u\n",i, *(HostMem->n_photons_left));
  }

  printf("[GPU] simulation done!\n");

  CopyDeviceToHostMem(HostMem, &DeviceMem, hstate->sim);
  FreeDeviceSimStates(&DeviceMem, &tstates);
  // We still need the host-side structure.
}

//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
static void DoOneSimulation(int sim_id, SimulationStruct* simulation,
                            unsigned long long *x, unsigned int *a)
{
  printf("\n------------------------------------------------------------\n");
  printf("        Simulation #%d\n", sim_id);
  printf("        - number_of_photons = %u\n", simulation->number_of_photons);
  printf("------------------------------------------------------------\n\n");

  // Start simulation kernel exec timer
  unsigned int execTimer = 0;
  CUT_SAFE_CALL( cutCreateTimer( &execTimer));
  CUT_SAFE_CALL( cutStartTimer(execTimer));

  //clock_t time1,time2;

  //// Start the clock
  //time1=clock();

  // For each GPU, init the host-side structure.
  HostThreadState* hstates;
  hstates = (HostThreadState*)malloc(sizeof(HostThreadState));

  hstates->sim = simulation;

  SimState *hss = &(hstates->host_sim_state);

  // number of photons responsible 
  hss->n_photons_left = (unsigned int*)malloc(sizeof(unsigned int));
  *(hss->n_photons_left) = simulation->number_of_photons; 

  // random number seeds
  hss->x = &x[0]; hss->a = &a[0];

  // Launch simulation
  RunGPUi (hstates);

  // End the timer.
  //time2=clock();
  //printf("\n*** Simulation time: %.3f sec\n\n",(double)(time2-time1)/CLOCKS_PER_SEC);
  CUT_SAFE_CALL( cutStopTimer(execTimer));
  printf( "\n\n>>>>>>Simulation time: %f (ms)\n", cutGetTimerValue(execTimer));
  
  Write_Simulation_Results(hss, simulation, cutGetTimerValue(execTimer));

  // Free SimState structs.
  FreeHostSimState(hss);
  free(hstates);
  CUT_SAFE_CALL( cutDeleteTimer( execTimer));
}

//////////////////////////////////////////////////////////////////////////////
//   Perform MCML simulation for one run out of N runs (in the input file)
//////////////////////////////////////////////////////////////////////////////
int main(int argc, char* argv[])
{
	//Init GPU Device
	CUT_DEVICE_INIT(argc, argv);

  char* filename = NULL;
  unsigned long long seed = (unsigned long long) time(NULL);
  int ignoreAdetection = 0;
  
  SimulationStruct* simulations;
  int n_simulations;

  int i;

  // Parse command-line arguments.
  if (interpret_arg(argc, argv, &filename,&seed, &ignoreAdetection))
  {
    usage(argv[0]);
    return 1;
  }

  // Output the execution configuration.
  printf("\n====================================\n");
  printf("EXECUTION MODE:\n");
  printf("  ignore A-detection:      %s\n", ignoreAdetection ? "YES" : "NO");
  printf("  seed:                    %llu\n", seed);
  printf("====================================\n\n");

  // Read the simulation inputs.
  n_simulations = read_simulation_data(filename, &simulations, ignoreAdetection);
  if(n_simulations == 0)
  {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  }
  printf("Read %d simulations\n",n_simulations);

  // Allocate and initialize RNG seeds.
  unsigned int len = NUM_THREADS;

  unsigned long long *x = (unsigned long long*)malloc(len * sizeof(unsigned long long));
  unsigned int *a = (unsigned int*)malloc(len * sizeof(unsigned int));

#ifdef _WIN32 
  if (init_RNG(x, a, len, "safeprimes_base32.txt", seed)) return 1;
#else 
  if (init_RNG(x, a, len, "executable/safeprimes_base32.txt", seed)) return 1;
#endif
  
  printf("Using the MWC random number generator ...\n");

  //perform all the simulations
  for(i=0;i<n_simulations;i++)
  {
    // Run a simulation
    DoOneSimulation(i, &simulations[i], x, a);
  }

  // Free the random number seed arrays.
  free(x); free(a);
  FreeSimulationStruct(simulations, n_simulations);

  return 0; 
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////